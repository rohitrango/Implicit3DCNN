#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <stdexcept>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define THREADS 512
#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define div_up(x, n) ((x)+(n)-1)/(n)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ int compute_ravel_hash(const int *coord, const int resolution, const int hashmap_size) {
    // compute hash function for tiled grid
    int index = 0;
    int stride = 1;
    #pragma unroll
    for(int i=0; i<3; i++) {
        index += coord[i]*stride;
        stride *= resolution;
    }
    return index % hashmap_size;
}

__device__ int get_level(const int* __restrict__  offsets, const int tableoffset, const int num_levels) {
    // given the tableoffset in the big hash table, get its level
    for(int i=1; i<num_levels; i++) {
        if(tableoffset < offsets[i]) {
            return i-1;
        }
    }
    return num_levels-1;
}

__device__ void unravel_index(const int index, const int resolution, int* coord) {
    // unravel index 
    int res2 = resolution*resolution;
    coord[2] = index / res2;
    coord[1] = (index%res2) / resolution;
    coord[0] = index % resolution;
}

template <typename scalar_t>
__global__ void abstract_contextlayer_forward_kernel(
    const scalar_t *input, 
    scalar_t *output,
    const int *offsets, 
    const int *resolutions,
    const int batch_size, 
    const int num_embeddings, 
    const int input_channels, 
    const int num_levels, 
    const int hashmap_size) 
{
    int num = batch_size*num_embeddings*input_channels;
    int output_channels = input_channels;
    int offset1 = offsets[1];

    CUDA_KERNEL_LOOP(index, num) {
        // this is at least the first level
        int c_idx = index % output_channels;
        int b_idx = (index / output_channels) % batch_size;
        int n_idx = (index / output_channels) / batch_size;
        // level related variables
        int level = get_level(offsets, n_idx, num_levels);
        // this is the first layer, no context from above
        if(level == 0)
            continue;
        int offset_lvl_prev = offsets[level - 1];
        int offset_lvl = offsets[level];
        int local_n = n_idx - offset_lvl;
        int lvl_res_prev = resolutions[level-1];
        int lvl_res = resolutions[level];
        int lvl_res3 = lvl_res*lvl_res*lvl_res;
        // if this is a tail end, skip t
        if(local_n >= lvl_res3)
            continue;
        // initialize the result
        scalar_t res = 0;
        // initialize the index here and iterate through all possible coordinates at this resolution 
        int _iter_local_n = local_n;
        while(_iter_local_n < lvl_res3) {
            int coord[3];
            unravel_index(_iter_local_n, lvl_res, coord); 
            float coord_f[3];
            #pragma unroll
            for(int i=0; i<3; i++) 
                coord_f[i] = ((float)coord[i]) / lvl_res * lvl_res_prev;
            // given the nearest float coordinates at the previous level 
            #pragma unroll
            for(int i=0; i<3; i++)
                coord[i] = (int)(coord_f[i] + 0.49);
            // this coordinate is in the previous level
            int xindex = compute_ravel_hash(coord, lvl_res_prev, hashmap_size) + offset_lvl_prev;
            res += input[xindex*batch_size*input_channels + b_idx*input_channels + c_idx];
            // get to the next size
            _iter_local_n += hashmap_size;
        }
        output[n_idx*batch_size*output_channels + b_idx*output_channels + c_idx] = res;
    }
}


template <typename scalar_t>
void abstract_contextlayer_forward_wrapper(const scalar_t *input, scalar_t *output, const int *offsets, const int *resolutions,
        const int batch_size, const int num_embeddings, const int input_channels, const int num_levels, const int hashmap_size) {
    
    const uint32_t blocks = min(div_up(num_embeddings*batch_size*input_channels, THREADS), 1<<30 - 1);
    abstract_contextlayer_forward_kernel<<<blocks, THREADS, 0,  at::cuda::getCurrentCUDAStream()>>>(
        input, output, offsets, resolutions, batch_size, num_embeddings, input_channels, num_levels, hashmap_size);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

torch::Tensor abstract_contextlayer_forward(torch::Tensor input, torch::Tensor output, torch::Tensor offsets, torch::Tensor resolutions,
                         int num_levels, int hashmap_size) {
    CHECK_CUDA(input);
    CHECK_CUDA(output);
    CHECK_CUDA(offsets);
    CHECK_CUDA(resolutions);
    CHECK_CONTIGUOUS(input);
    CHECK_CONTIGUOUS(output);
    CHECK_CONTIGUOUS(offsets);
    CHECK_CONTIGUOUS(resolutions);

    if(input.dim() != 3) {
        throw std::runtime_error("Input must have 3 dimensions");
    }
    // define extra variables
    const int num_embeddings = input.size(0);
    const int batch_size = input.size(1);
    const int input_channels = input.size(2);
    if(input.size(2) != output.size(2)) {
        throw std::runtime_error("Input and output must have same sizes");
    }
    // kernel sizes (first index is levels)

    AT_DISPATCH_FLOATING_TYPES( 
    input.scalar_type(), "abstract_contextlayer_forward_wrapper", ([&] {
        abstract_contextlayer_forward_wrapper<scalar_t>(input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), offsets.data_ptr<int>(), resolutions.data_ptr<int>(), 
            batch_size, num_embeddings, input_channels, num_levels, hashmap_size);
    }));
    return output;
}

torch::Tensor abstract_contextlayer_backward(torch::Tensor grad_output, torch::Tensor grad_input, 
        torch::Tensor offsets, torch::Tensor resolutions, int num_levels, int hashmap_size) {
    // TODO: Not implemented yet
    return grad_input;
}